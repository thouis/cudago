
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdio.h>
#include <assert.h>

#define LOG 0

// We want to be able to hold a board in shared memory

#define b00 0
#define b01 1
#define b10 2
#define b11 3

// Board values
#define EMPTY 0
#define BLACK 1
#define WHITE 2
#define EDGE  3
#define BLACK_ALIVE 4
#define WHITE_ALIVE 5

#define ALIVE_OFFSET 3

const char stone_chars[] = ".#o ";

#define NAME(color) ((color) == WHITE ? "white" : "black")

typedef struct board {
    struct col {
        // Each col & row is 21 entries wide to allow for edges.
        // Values defined above
        uint32_t rows[21];
    };
    col cols[21];
    uint8_t flags;
    uint8_t ko_row, ko_col;
} Board;

#define STONE_AT(b, r, c) ((b)->cols[c].rows[r])
#define SET_STONE_AT(b, r, c, v) ((b)->cols[c].rows[r] = v)

#define OPPOSITE(color) ((color == WHITE) ? BLACK : WHITE)

// XXX - TODO:
// - write scoring function.
// - ko detection - testing.
// - is there an easy way to avoid playing in single eyes that are controlled by other player?
//    Maybe:
//        add WHITE_PERMANENT and BLACK_PERMANENT: if alive and has two real single-space eyes.
//        don't play in single spaces surrounded by PERMANENTs.


#define IS_NEXT_TO(b, r, c, v)  ((STONE_AT(b, r + 1, c) == (v)) || \
                                 (STONE_AT(b, r - 1, c) == (v)) || \
                                 (STONE_AT(b, r, c + 1) == (v)) || \
                                 (STONE_AT(b, r, c - 1) == (v)))

#define CT_NEXT_TO(b, r, c, v)  ((STONE_AT(b, r + 1, c) == (v)) + \
                                 (STONE_AT(b, r - 1, c) == (v)) + \
                                 (STONE_AT(b, r, c + 1) == (v)) + \
                                 (STONE_AT(b, r, c - 1) == (v)))


// SINGLE EYE == 4 horizontal & vertical neighbors are all the right color, or edge.
#define SINGLE_EYE(b, r, c, color) ((CT_NEXT_TO(b, r, c, color) + CT_NEXT_TO(b, r, c, EDGE)) == 4)


// FALSE_EYE is only valid if SINGLE_EYE is true
// - >= two diagonal neighbors are opposite color, or
// - 1 diagonal neighbor opposite color and at edge
#define DIAG_NEIGHBORS(b, r, c, color) (((STONE_AT(b, r + 1, c + 1) == color) ? 1 : 0) + \
                                        ((STONE_AT(b, r + 1, c - 1) == color) ? 1 : 0) + \
                                        ((STONE_AT(b, r - 1, c + 1) == color) ? 1 : 0) + \
                                        ((STONE_AT(b, r - 1, c - 1) == color) ? 1 : 0))

#define AT_EDGE(b, r, c) IS_NEXT_TO(b, r, c, EDGE)

#define FALSE_EYE(b, r, c, color)  ((DIAG_NEIGHBORS(b, r, c, OPPOSITE(color)) >= 2) || \
                                    ((DIAG_NEIGHBORS(b, r, c, OPPOSITE(color)) == 1) && AT_EDGE(b, r, c)))


// Real single eyes = single eye, and not false.  Fails for some cases
// (see Two-headed dragon @ sensei's library).
#define SINGLE_REAL_EYE(b, r, c, color) (SINGLE_EYE(b, r, c, color) && (! FALSE_EYE(b, r, c, color)))

#define ALIVE(b, row, c, alive_color) (IS_NEXT_TO(b, row, c, EMPTY) || IS_NEXT_TO(b, row, c, alive_color))

#define LONE_ATARI(b, row, c, color) ((CT_NEXT_TO(b, row, c, EMPTY) == 1) && (! (IS_NEXT_TO(b, row, c, color))))


__global__ void clear_board(Board *b)
{
    int row = threadIdx.x;
    if (row < 21) {
        SET_STONE_AT(b, row, 0, EDGE);
        for (int c = 1; c <= 19; c++)
            SET_STONE_AT(b, row, c, ((row == 0) || (row == 20)) ? EDGE : EMPTY);
        SET_STONE_AT(b, row, 20, EDGE);
    }
    if (row == 0) {
        b->flags = 0;
        b->ko_row = 0;
    }
}

__device__ int remove_dead_groups(Board *b,
                                  uint8_t color)
{
    // NB: we add because interior space on board is 1 indexed
    int row = threadIdx.x + 1;
    int num_changes;
    int alive_color = color + ALIVE_OFFSET;

    // Loop until no new updates have been made
    num_changes = 1;
    while (num_changes > 0) {
        num_changes = 0;
        for (int c = 1; c <= 19; c++) {
            if ((STONE_AT(b, row, c) == color) && ALIVE(b, row, c, alive_color)) {
                SET_STONE_AT(b, row, c, alive_color);
                num_changes++;
            }
        }
        // figure out how many updates there were on the whole board
        num_changes += __shfl_down(num_changes, 16);
        num_changes += __shfl_down(num_changes, 8);
        num_changes += __shfl_down(num_changes, 4);
        num_changes += __shfl_down(num_changes, 2);
        num_changes += __shfl_down(num_changes, 1);

        // update all threads about total updates
        num_changes = __shfl(num_changes, 0);
    }

    // replace alive stones with stones of that color, and not-alive with empty.
    num_changes = 0;
    for (int c = 1; c <= 19; c++) {
        if (STONE_AT(b, row, c) == color) {
            SET_STONE_AT(b, row, c, EMPTY);
            num_changes++;
        }
        else if (STONE_AT(b, row, c) == alive_color)
            SET_STONE_AT(b, row, c, color);
    }

    num_changes += __shfl_down(num_changes, 16);
    num_changes += __shfl_down(num_changes, 8);
    num_changes += __shfl_down(num_changes, 4);
    num_changes += __shfl_down(num_changes, 2);
    num_changes += __shfl_down(num_changes, 1);

    // update all threads about total removed
    num_changes = __shfl(num_changes, 0);

    if (LOG && (row == 1)) {
        if (num_changes > 0)
            printf("    removed %d %s stones\n", num_changes, NAME(color));
    }

    // NB: all threads must return the same value for
    // make_random_move() to work correctly.
    return num_changes;
}

// **************************************************
// Makes a random move and returns true if the board changed.
// **************************************************
__device__ int make_random_move(Board *b,
                                uint8_t color,
                                hiprandState *randstate)
{
    // NB: we add because interior space on board is 1 indexed
    int row = threadIdx.x + 1;

    // local values
    int num_valid_moves;
    int valid_move_mask;
    int num_killed;
    int num_suicide;

    // where the random move is made
    int which_move = 0;
    int which_row;
    int which_col;

    // shared values
    __shared__ int thread_valid_moves[20];


    // remember 1-indexed because of edges, and see NB above
    if (row > 19) return 0;

    // **************************************************
    // FIND VALID PLAY LOCATIONS
    // **************************************************
    valid_move_mask = 0;
    for (int c = 1; c <= 19; c++) {
        if ((STONE_AT(b, row, c) == EMPTY) && (! SINGLE_REAL_EYE(b, row, c, color))) {
            valid_move_mask |= (1 << c);
        }
    }

    // Disallow retaking the ko
    if (row == b->ko_row) {
        valid_move_mask &= ~ (1 << b->ko_col);
    }

    num_valid_moves = __popc(valid_move_mask);
    thread_valid_moves[row] = num_valid_moves;

    // figure out how many valid moves there were in the whole board
    num_valid_moves += __shfl_down(num_valid_moves, 16);
    num_valid_moves += __shfl_down(num_valid_moves, 8);
    num_valid_moves += __shfl_down(num_valid_moves, 4);
    num_valid_moves += __shfl_down(num_valid_moves, 2);
    num_valid_moves += __shfl_down(num_valid_moves, 1);

    // update all threads about valid move count
    num_valid_moves = __shfl(num_valid_moves, 0);

    if (num_valid_moves == 0) {
        // forced pass

        // clear ko flag
        if (row == 1)
            b->ko_row = 0;
        return 0;  // no change in board
    }


    // **************************************************
    // CHOOSE RANDOM ROW BASED ON VALID MOVE COUNTS
    // **************************************************
    if (row == 1) {
        // choose one row to execute a move
        which_move = hiprand(randstate) % num_valid_moves;
        which_row = 1;
        while (which_move >= thread_valid_moves[which_row]) {
            which_move -= thread_valid_moves[which_row];
            which_row++;
        }
    }

    // all threads have to execute the shuffle
    valid_move_mask = __shfl(valid_move_mask, which_row - 1);

    // **************************************************
    // MAKE RANDOM MOVE IN CHOSEN ROW
    // **************************************************
    if (row == 1) {
        // find which column to place move at
        which_col = 1;
        do {
            // shift which_col to the next set bit in valid_move_mask
            while (! (valid_move_mask & (1 << which_col)))
                which_col++;
            if (which_move > 0)
                which_col++;
            which_move--;
        } while (which_move >= 0);
        if (LOG) {
            printf("%d total valid moves\n", num_valid_moves);
            printf("    placed %s at %d %d\n", NAME(color), which_row, which_col);
        }
        SET_STONE_AT(b, which_row, which_col, color);
    }

    // update all threads about where we played
    which_row = __shfl(which_row, 0);
    which_col = __shfl(which_col, 0);
    
    // **************************************************
    // REMOVE DEAD GROUPS & KO DETECTION
    // **************************************************
    num_killed = 0;
    num_suicide = 0;

    if (IS_NEXT_TO(b, which_row, which_col, OPPOSITE(color)))
        num_killed = remove_dead_groups(b, OPPOSITE(color));

    // only check for suicide moves if necessary
    if ((num_killed == 0) && (! IS_NEXT_TO(b, which_row, which_col, EMPTY)))
        num_suicide = remove_dead_groups(b, color);
        

    if (row == 1) {
        if ((num_killed == 1) && LONE_ATARI(b, which_row, which_col, color)) {
            if      (STONE_AT(b, which_row + 1, which_col) == EMPTY) { b->ko_row = which_row + 1; b->ko_col = which_col; }
            else if (STONE_AT(b, which_row - 1, which_col) == EMPTY) { b->ko_row = which_row - 1; b->ko_col = which_col; }
            else if (STONE_AT(b, which_row, which_col + 1) == EMPTY) { b->ko_row = which_row;     b->ko_col = which_col + 1; }
            else if (STONE_AT(b, which_row, which_col - 1) == EMPTY) { b->ko_row = which_row;     b->ko_col = which_col - 1; }
            if (LOG) printf("     ko at %d %d\n", b->ko_row, b->ko_col);
         } else {
            b->ko_row = b->ko_col = 0;
        }
    }

    // NB: all threads will return the same value

    // Return whether the board changed.
    // The only way that didn't happen is if this was a single-stone suicide play.
    return (num_suicide != 1);
}

__global__ void play_out(Board *start_board,
                         Board *boards,
                         uint8_t first_move_color,
                         int max_moves,
                         int max_unchanged,
                         hiprandState *randstates)
{
    int move_count = 0;
    int unchanged_count = 0;
    uint8_t current_color = first_move_color;
    hiprandState *my_rand = randstates + blockIdx.x;
    Board *my_board = boards + blockIdx.x;

    // NB: we add because interior space on board is 1 indexed
    int row = threadIdx.x + 1;

    if (row > 19) return;

    if (row == 1)
        *my_board = *start_board;

    while ((move_count < max_moves) && (unchanged_count < max_unchanged)) {
        move_count++;
        int board_changed = make_random_move(my_board, current_color, my_rand);
        unchanged_count = board_changed ? 0 : (unchanged_count + 1);
        current_color = OPPOSITE(current_color);
        if (LOG && (row == 1))
            printf("unchanged: %d, move_count: %d\n", unchanged_count, move_count);
    }
}

__global__ void sum_boards(Board *start_board,
                           int num_boards,
                           Board *dest_board)
{
    int row = threadIdx.x + 1;
    int col = threadIdx.y + 1;
    if ((row > 19) || (col > 19))
        return;

    int count = 0;

    for (int i = 0; i < num_boards; i++) {
        int color = STONE_AT(start_board + i, row, col);
        if ((color == BLACK) || ((color == EMPTY) &&
                                 IS_NEXT_TO(start_board + i, row, col, BLACK)))
            count++;
    }
    SET_STONE_AT(dest_board, row, col, count);
}

__global__ void setup_random(hiprandState *states)
{
    unsigned int id = blockIdx.x;
    unsigned int seed = (unsigned int) clock64();
    hiprand_init(seed ^ (id << 6), id, 0, &(states[id]));
}

#define COUNT 10000

int main(void)
{
    void *start_board, *playouts, *board_sum;
    Board board;
    hiprandState *randstates;
    hipEvent_t start, end;

    hipMalloc(&start_board, sizeof (Board));
    hipMalloc(&playouts, COUNT * sizeof (Board));
    hipMalloc(&board_sum, sizeof (Board));
    hipMalloc(&randstates, COUNT * sizeof(hiprandState));

    hipEventCreate(&start);
    hipEventCreate(&end);

    setup_random<<<COUNT, 1>>>(randstates);
    clear_board<<<1, 32>>>((Board *) start_board);
    hipMemcpy(&board, (Board *) start_board, sizeof (Board), hipMemcpyDeviceToHost);

    // Game #768554 - played out by gnugo.  3.5 at the end in chinese scoring.
    char game[] = "[pd][dp][pq][dd][qk][lp][cj][cl][cg][gc][lc][jc][le][qc][qd][pc][nc][od][oc][rd][re][rc][qe][oe][qg][og][lg][oi][qi][ok][cn][dk][cq][cp][dq][ep][eq][fq][fr][gq][gr][hq][bp][bo][bq][bn][be][nq][po][li][pb][qb][ob][kh][mn][on][np][op][oo][no][mp][mo][mq][mr][nn][lo][om][pl][pp][lq][ln][ql][pm][kn][km][jm][jl][kl][lm][im][jk][lk][hk][ih][fk][gl][gk][fl][fh][ej][gi][kg][ek][dj][el][dh][dg][eg][eh][ci][ef][ge][gf][fe][if][ff][fg][hf][hg][gg][hh][ig][cc][dc][cb][kb][lb][lf][mf][kf][mg][me][ne][md][nd][nf][ke][mh][ng][of][ld][nh][mc][em][dl][dm][id][ic][rk][je][jd][ie][hd][cd][bd][qm][qn][hc][kc][me][md][ja][sb][hr][er][or][pr][ps][qs][os][qr][sm][rn][db][da][ea][ca][fb][rl][rm][sn][bh][bg][hl][il][jn][ch][bj][di][bi][hs][is][gs][ij][ik][ir][la][lh][me][jj][nk][nj][mk][mj][ol][pk][pj][pi][ph][oh][qj][oj][ll][kk][ce][hj][gj][hi][gh][pg][qh][ag][af][ah][ee][ed][sl][ap][aq][ao][na][gd][ei][de][df][qf][rf][pf][ka][jb][fj][ck][oq][np][gf][he][kd][pa][mb][co]";
    int curcolor = BLACK;
    for (int idx = 1; idx < sizeof(game); idx += 4) {
        int c = game[idx] - 'a' + 1;
        int r = game[idx + 1] - 'a' + 1;
        SET_STONE_AT(&board, r, c, curcolor);
        curcolor = OPPOSITE(curcolor);
    }
    hipMemcpy((Board *) start_board, &board, sizeof (Board), hipMemcpyHostToDevice);
    
    
    hipEventRecord(start, 0);
    play_out<<<COUNT, 32>>>((Board *) start_board, (Board *) playouts, 
                            BLACK, 1000, 100, randstates);

    sum_boards<<<1, dim3(19, 19)>>>((Board *) playouts, COUNT, (Board *) board_sum);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float delta_ms;
    hipEventElapsedTime(&delta_ms, start, end);
    printf("%d boards in %0.2f ms\n", COUNT, delta_ms);


    hipMemcpy(&board, ((Board *) playouts), sizeof (Board), hipMemcpyDeviceToHost);
    for(int i = 0; i < 21; i++) {
        for(int j = 0; j < 19; j++) {
            printf("%c ", stone_chars[STONE_AT(&board, i, j)]);
        }
        printf("\n");
    }
    printf("\n");


    hipMemcpy(&board, ((Board *) board_sum), sizeof (Board), hipMemcpyDeviceToHost);

    int total = 0;

    printf("[");
    for(int i = 0; i < 19; i++) {
        printf ("[");
        for(int j = 0; j < 19; j++) {
            printf("%d,", STONE_AT(&board, i + 1, j + 1));
            total += STONE_AT(&board, i + 1, j + 1);
        }
        printf("],\n");
    }
    printf("]\n");
    printf("expected score (B over W): %f\n", 2 * (total / ((float) COUNT)) - 361.0);

    hipDeviceReset();
    return 0;
}
